
#include <hip/hip_runtime.h>
#include <stdio.h>

template<unsigned int warpsize>
__global__ void what_is_my_id(unsigned int * block,
                         unsigned int * thread, 
                         unsigned int * warp, 
                         unsigned int * calc_thread ){
    const unsigned int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    block[thread_idx] = blockIdx.x;
    thread[thread_idx] = threadIdx.x;

    warp[thread_idx] = threadIdx.x / warpsize;
    calc_thread[thread_idx] = thread_idx;
}

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * ARRAY_SIZE)

unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];

int main(){
    const unsigned int num_blocks = 2;
    const unsigned int num_threads = 64;
    const unsigned int warp_size = 32;

    unsigned int * gpu_block;
    unsigned int * gpu_thread;
    unsigned int * gpu_warp;
    unsigned int * gpu_calc_thread;

    unsigned int i;
    
    hipMalloc((void**)&gpu_block, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_warp, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void**)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);

    what_is_my_id<warp_size><<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);

    hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    hipFree(gpu_block);
    hipFree(gpu_thread);
    hipFree(gpu_warp);
    hipFree(gpu_calc_thread);

    for(i=0; i< ARRAY_SIZE; i++){
        printf("cac_thread %3u - block %2u - warp %3u - thread %2u\n", cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
    }

}
